// #pragma GCC optimize(2)

#include <hip/hip_runtime.h>

#include <iostream>

void helloFromCPU()
{
    printf("Hello World from CPU!\n");
}

__global__ void helloFromGPU()
{
    printf("Hello World from GPU!\n");
}

__global__ void gpu_print(int N){
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    if(idx<N){
        printf("%d\n",idx);
    }
}
int main() {
    // helloFromGPU <<<1, 2>>> ();
    // // cudaDeviceSynchronize();//同步函数，阻塞接下来串行执行的cpu任务
    // helloFromCPU();

    gpu_print<<<6,4>>>(24);

    
    hipDeviceReset();
    return 0;
}
